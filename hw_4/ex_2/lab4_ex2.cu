#include "hip/hip_runtime.h"
#include <stdio.h>
#include <random>
#include <hip/hip_runtime.h>

#define DataType double
#define TPB 256

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len) {
  //@@ Insert code to implement vector addition here
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < len) {
        out[i] = in1[i] + in2[i];
    }
}


int main(int argc, char **argv) {

    int inputLength;
    int S_seg;
    DataType *hostInput1;
    DataType *hostInput2;
    DataType *hostOutput;
    DataType *resultRef;
    DataType *deviceInput1;
    DataType *deviceInput2;
    DataType *deviceOutput;

  //@@ Insert code below to read in inputLength from args
    inputLength = atoi(argv[1]);
    S_seg = atoi(argv[2]);

    printf("The input length is %d\n", inputLength);
    printf("The segment size is %d\n", S_seg);

    hipStream_t streams[4];
    for (int i = 0; i < 4; i++) {
        hipStreamCreate(&streams[i]);
    }

    //@@ Insert code below to allocate Host memory for input and output
    hipHostMalloc((void **)&hostInput1, inputLength * sizeof(DataType));
    hipHostMalloc((void **)&hostInput2, inputLength * sizeof(DataType));
    hipHostMalloc((void **)&hostOutput, inputLength * sizeof(DataType));
    hipHostMalloc((void **)&resultRef, inputLength * sizeof(DataType));

    //@@ Insert code below to initialize hostInput1 and hostInput2 to random numbers, and create reference result in CPU
    for (int i = 0; i < inputLength; i++) {
        hostInput1[i] = (DataType)rand() / (DataType)RAND_MAX;
        hostInput2[i] = (DataType)rand() / (DataType)RAND_MAX;
        resultRef[i] = hostInput1[i] + hostInput2[i];
    }

    //@@ Insert code below to allocate GPU memory here
    hipMalloc((void **)&deviceInput1, inputLength * sizeof(DataType));
    hipMalloc((void **)&deviceInput2, inputLength * sizeof(DataType));
    hipMalloc((void **)&deviceOutput, inputLength * sizeof(DataType));

    int gridDim = (S_seg/ TPB);

    int iterations = inputLength / S_seg + 1;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    for (int i = 0; i < iterations; i++){
        int stream = i % 4;
        int offset = i * S_seg;
        int length = min(S_seg, inputLength - offset);
        //printf("The length is %d\n", length);
        //printf("S_seg is %d\n", S_seg);
        int bytes = min(S_seg, inputLength - offset) * sizeof(DataType);

        hipMemcpyAsync(deviceInput1 + offset, hostInput1 + offset, bytes, hipMemcpyHostToDevice, streams[stream]);
        hipMemcpyAsync(deviceInput2 + offset, hostInput2 + offset, bytes, hipMemcpyHostToDevice, streams[stream]);
        vecAdd<<<gridDim, TPB, 0, streams[stream]>>>(deviceInput1 + offset, deviceInput2 + offset, deviceOutput + offset, length);
        hipMemcpyAsync(hostOutput + offset, deviceOutput + offset, bytes, hipMemcpyDeviceToHost, streams[stream]);
    }
    for (int i = 0; i < 4; i++) {
        hipStreamSynchronize(streams[i]);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Execution time is %f ms\n", milliseconds);

    //@@ Insert code below to compare the output with the reference
    int comparisonCheck = 1;
    for (int i = 0; i < inputLength; i++) {
        if (hostOutput[i] - resultRef[i] > 1e-6) {
            comparisonCheck = 0;
            break;
        }
    }

    if (comparisonCheck == 1) {
        printf("The comparison is equal!\n");
    } else {
        printf("The comparison is not equal.\n");
    }

    for (int i = 0; i < 4; i++) {
        hipStreamDestroy(streams[i]);
    }

    hipFree(deviceInput1);
    hipFree(deviceInput2);
    hipFree(deviceOutput);

    hipFree(hostInput1);
    hipFree(hostInput2);
    hipFree(hostOutput);
    hipFree(resultRef);

    return 0;
}
